
#include <hip/hip_runtime.h>
// Kernel algorithm for temperature computation
__global__ void compute_temperature(double* T, double* T_new, double* q, double k, 
    int grid_size, double h, double T_amb) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int shared_x = threadIdx.x + 1; // +1 for halo
    int shared_y = threadIdx.y + 1; 

    __shared__ double s_T[18][19]; // (blockDim.x + 2 x blockDim.y + 2) including halos

    // Boundary check and load center cell
    if (x < grid_size && y < grid_size) {
        s_T[shared_y][shared_x] = T[y * grid_size + x];
    } else {
        s_T[shared_y][shared_x] = T_amb;
    }


    // Load halos: left
    if (threadIdx.x == 0) {
        int halo_x = x - 1;
        int halo_y = y;
        s_T[shared_y][0] = (halo_x >= 0 && halo_y < grid_size) ? T[halo_y * grid_size + halo_x] : T_amb;
    }
    // Right halo
    if (threadIdx.x == 15) {
        int halo_x = x + 1;
        int halo_y = y;
        s_T[shared_y][17] = (halo_x < grid_size && halo_y < grid_size) ? T[halo_y * grid_size + halo_x] : T_amb;
    }
    // Top halo
    if (threadIdx.y == 0) {
        int halo_x = x;
        int halo_y = y - 1;
        s_T[0][shared_x] = (halo_y >= 0 && halo_x < grid_size) ? T[halo_y * grid_size + halo_x] : T_amb;
    }
    // Bottom halo
    if (threadIdx.y == 15) {
        int halo_x = x;
        int halo_y = y + 1;
        s_T[17][shared_x] = (halo_y < grid_size && halo_x < grid_size) ? T[halo_y * grid_size + halo_x] : T_amb;
    }

    __syncthreads();

    if (x >= grid_size || y >= grid_size) return; // bounds check

    int idx = y * grid_size + x;

    // Apply Dirichlet boundary conditions
    if (x == 0 || x == grid_size - 1 || y == 0 || y == grid_size - 1) {
        T_new[idx] = T_amb;
        return;
    }

    // Load values for neighbors
    double top = s_T[shared_y - 1][shared_x];
    double bottom = s_T[shared_y + 1][shared_x];
    double left = s_T[shared_y][shared_x - 1];
    double right = s_T[shared_y][shared_x + 1];

    double coeff = (h * h / k) * q[idx];

    T_new[idx] = (top + bottom + left + right + coeff) / 4.0;
}

// Kernel for reduction to find maximum difference
__global__ void max_diff_reduction(double* T, double* T_new, double* max_diff, int total_size) {
    __shared__ double data[256];
    int local_index = threadIdx.y * blockDim.x + threadIdx.x;
    int global_index = blockIdx.x * blockDim.x * blockDim.y + local_index;


    // Compute difference for each thread
    double difference = 0.0;
    if (global_index < total_size) {
        difference = fabs(T_new[global_index] - T[global_index]);
    }

    data[local_index] = difference;
    __syncthreads();

    // Max reduction
    for (int stride = 128; stride > 0; stride /= 2) {
        if (local_index  < stride) {
            data[local_index] = fmax(data[local_index], data[local_index + stride]);
        }
        __syncthreads();
    }

    // Return the maximum difference at index 0
    if (local_index  == 0) {
        max_diff[blockIdx.x] = data[0];
    }
}


// Compute the maximum, minimum, and average temperature in the grid
double max_temp(double* arr, int grid_size) {
    double max_val = arr[0];
    for (int i = 0; i < grid_size * grid_size; i++) {
        if (arr[i] > max_val) max_val = arr[i];
    }
    return max_val;  
}
double min_temp(double* arr, int grid_size) {
    double min_val = arr[0];
    for (int i = 0; i < grid_size * grid_size; i++) {
        if (arr[i] < min_val) min_val = arr[i];
    }
    return min_val;
}
double avg_temp(double* arr, int grid_size) {
    double sum = 0.0;
    for (int i = 0; i < grid_size * grid_size; i++) {
        sum += arr[i];
    }
    return (sum / (grid_size * grid_size));
}

