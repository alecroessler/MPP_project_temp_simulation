
#include <hip/hip_runtime.h>




__global__ void compute_temperature(double* T, double* T_new, double* q, double k, 
    int grid_size, double h, double T_amb) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= grid_size || y >= grid_size) return;

    int idx = y * grid_size + x;

    // Apply Dirichlet boundary conditions
    if (x == 0 || x == grid_size - 1 || y == 0 || y == grid_size - 1) {
        T_new[idx] = T_amb;
        return;
    }

    // Compute 1D indices for neighbors
    int top    = (y - 1) * grid_size + x;
    int bottom = (y + 1) * grid_size + x;
    int left   = y * grid_size + (x - 1);
    int right  = y * grid_size + (x + 1);

    double coeff = (h * h / k) * q[idx];

    T_new[idx] = (T[top] + T[bottom] + T[left] + T[right] + coeff) / 4.0;
}


__global__ void compute_diff(double* T, double* T_new, double* diff, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        diff[idx] = fabs(T[idx] - T_new[idx]);
    }
}

__global__ void reduce_max(double* input, double* output, int size) {
    extern __shared__ double sdata[];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    double max_val = 0.0;

    // Load elements into shared memory (2 elements per thread to optimize)
    if (idx < size)
        max_val = input[idx];
    if (idx + blockDim.x < size) {
        double other = input[idx + blockDim.x];
        if (other > max_val)
            max_val = other;
    }

    sdata[tid] = max_val;
    __syncthreads();

    // Do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (sdata[tid + s] > sdata[tid])
                sdata[tid] = sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the block's maximum to output array
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}




double max_abs_diff(double* a, double* b, int size) {
    double max_diff = 0.0;
    for (int i = 0; i < size; ++i) {
        double diff = fabs(a[i] - b[i]);
        if (diff > max_diff) max_diff = diff;
    }
    return max_diff;
}



double max_temp(double* arr, int grid_size) {
    double max_val = arr[0];
    for (int i = 0; i < grid_size * grid_size; i++) {
        if (arr[i] > max_val) max_val = arr[i];
    }
    return max_val;  
}


double min_temp(double* arr, int grid_size) {
    double min_val = arr[0];
    for (int i = 0; i < grid_size * grid_size; i++) {
        if (arr[i] < min_val) min_val = arr[i];
    }
    return min_val;
}


double avg_temp(double* arr, int grid_size) {
    double sum = 0.0;
    for (int i = 0; i < grid_size * grid_size; i++) {
        sum += arr[i];
    }
    return (sum / (grid_size * grid_size));
}

